#include "hip/hip_runtime.h"
#include "hiprand/hiprand_kernel.h"
#include "sem.h"
#include "stg_inputs.h"
#include "stg_utils.h"
#include <fstream>
#include <vector>

template class SEM<double>;

template <typename T> __device__ T f_1d_gpu(const T x) {
  T _f1d;
  if (std::abs(x) < 1.0)
    _f1d = std::sqrt(1.5) * (1.0 - std::abs(x));
  else
    _f1d = 0.0;
  return _f1d;
}

__global__ void setup_kernel(hiprandState *state) {

  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  /* Each thread gets same seed , a different sequence
  number , no offset */
  hiprand_init(1234, idx, 0, &state[idx]);
}

template <typename T>
__global__ void update_sem_gpu(T *xsem, T *ysem, T *zsem, T *eps_sem_x,
                               T *eps_sem_y, T *eps_sem_z, const T ub0,
                               const T dt, const T sigma_max, const T ly_sem,
                               const T lz_sem, hiprandState *state,
                               const int nsem) {

  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = gridDim.x * blockDim.x;
  /* Copy state to local memory for efficiency */
  hiprandState localState = state[idx];
  T rnd[5];

  for (int i = idx; i < nsem; i += stride) {
    xsem[i] += dt * ub0;
    if (xsem[i] > sigma_max) {

      for (auto k = 0; k < 5; ++k)
        rnd[k] = hiprand_uniform(&localState);
      xsem[i] = -sigma_max;
      ysem[i] = -sigma_max + rnd[0] * ly_sem;
      zsem[i] = -sigma_max + rnd[1] * lz_sem;

      eps_sem_x[i] = rnd[2] < 0.5 ? -1.0 : 1.0;
      eps_sem_y[i] = rnd[3] < 0.5 ? -1.0 : 1.0;
      eps_sem_z[i] = rnd[4] < 0.5 ? -1.0 : 1.0;
    }
  }
}

// Using atomic reduction
template <typename T>
__global__ void get_sem_fluc_gpu(const T *xsem, const T *ysem, const T *zsem,
                                 const T *eps_sem_x, const T *eps_sem_y,
                                 const T *eps_sem_z, T *uvec, const T sigma,
                                 const T *xx, const T *a_ij, const int nsem) {

  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = gridDim.x * blockDim.x;
  T usem = 0.0, vsem = 0.0, wsem = 0.0;

  for (int i = idx; i < nsem; i += stride) {
    T f1 = f_1d_gpu((xx[0] - xsem[i]) / sigma);
    T f2 = f_1d_gpu((xx[1] - ysem[i]) / sigma);
    T f3 = f_1d_gpu((xx[2] - zsem[i]) / sigma);

    T s1 = (a_ij[0] * eps_sem_x[i] + a_ij[1] * eps_sem_y[i] +
            a_ij[2] * eps_sem_z[i]);
    T s2 = (a_ij[3] * eps_sem_x[i] + a_ij[4] * eps_sem_y[i] +
            a_ij[5] * eps_sem_z[i]);
    T s3 = (a_ij[6] * eps_sem_x[i] + a_ij[7] * eps_sem_y[i] +
            a_ij[8] * eps_sem_z[i]);

    usem = usem + f1 * f2 * f3 * s1;
    vsem = vsem + f1 * f2 * f3 * s2;
    wsem = wsem + f1 * f2 * f3 * s3;
  }

  atomicAdd(&uvec[0], usem);
  atomicAdd(&uvec[1], vsem);
  atomicAdd(&uvec[2], wsem);
}

namespace Random {
std::mt19937_64 mt{std::random_device{}()};
template <typename T> void getrand(std::vector<T> &arr) {
  std::uniform_real_distribution<T> unif{0, 1};
  for (std::size_t i = 0; i < arr.size(); ++i)
    arr[i] = unif(mt);
}
} // namespace Random

template <typename T> SEM<T>::SEM(STGInterface<T> &stg_int) {
  T sigma_max = stg_int.get_sigma_max();
  lx_sem = STGInputs::lx + 2.0 * sigma_max;
  ly_sem = STGInputs::ly + 2.0 * sigma_max;
  lz_sem = STGInputs::lz + 2.0 * sigma_max;

  auto vol_sem = lx_sem * ly_sem * lz_sem;
  nsem = int(vol_sem * stg_int.get_sigma_inv3());

  // Assign the vectors
  xsem.assign(nsem, 0.0);
  ysem.assign(nsem, 0.0);
  zsem.assign(nsem, 0.0);

  eps_sem_x.assign(nsem, 0.0);
  eps_sem_y.assign(nsem, 0.0);
  eps_sem_z.assign(nsem, 0.0);

  std::cout << "Sigma max: " << sigma_max << std::endl;
  if (STGInputs::lGenerateSEM) {
      std::cout << "Generating SEM" << std::endl;
    generate_sem(stg_int);
   } else
    read_sem();
  // CUDA initialization
  size_t size = nsem * sizeof(T);

  hipMalloc((void **)&_gpu.xsem, size);
  hipMalloc((void **)&_gpu.ysem, size);
  hipMalloc((void **)&_gpu.zsem, size);

  hipMalloc((void **)&_gpu.uvec, 3 * sizeof(T));

  hipMalloc((void **)&_gpu.eps_sem_x, size);
  hipMalloc((void **)&_gpu.eps_sem_y, size);
  hipMalloc((void **)&_gpu.eps_sem_z, size);

  hipMalloc((void **)&_gpu.xx, 3 * sizeof(T));
  hipMalloc((void **)&_gpu.a_ij, 9 * sizeof(T));
  hipMemcpy(_gpu.xsem, xsem.data(), size, hipMemcpyHostToDevice);
  hipMemcpy(_gpu.ysem, ysem.data(), size, hipMemcpyHostToDevice);
  hipMemcpy(_gpu.zsem, zsem.data(), size, hipMemcpyHostToDevice);

  hipMemcpy(_gpu.eps_sem_x, eps_sem_x.data(), size, hipMemcpyHostToDevice);
  hipMemcpy(_gpu.eps_sem_y, eps_sem_y.data(), size, hipMemcpyHostToDevice);
  hipMemcpy(_gpu.eps_sem_z, eps_sem_z.data(), size, hipMemcpyHostToDevice);

  _gpu.sigma = 0.0;
}

template <typename T> void SEM<T>::generate_sem(STGInterface<T> &stg_int) {
  sigma_max = stg_int.get_sigma_max();

  Random::getrand(xsem);
  Random::getrand(ysem);
  Random::getrand(zsem);

  Random::getrand(eps_sem_x);
  Random::getrand(eps_sem_y);
  Random::getrand(eps_sem_z);

  for (auto i = 0; i < nsem; ++i) {
    xsem[i] = -sigma_max + xsem[i] * lx_sem;
    ysem[i] = -sigma_max + ysem[i] * ly_sem;
    zsem[i] = -sigma_max + zsem[i] * lz_sem;

    eps_sem_x[i] = eps_sem_x[i] < 0.5 ? -1.0 : 1.0;
    eps_sem_y[i] = eps_sem_y[i] < 0.5 ? -1.0 : 1.0;
    eps_sem_z[i] = eps_sem_z[i] < 0.5 ? -1.0 : 1.0;
  }
  write_sem();
}

template <typename T> void SEM<T>::write_sem() {
  std::string fname = "sem.dat";
  std::fstream f;
  f.open(fname, std::ios::out);
  if (f.is_open()) {
    f << nsem << "\n";
    f.precision(6);
    for (auto i = 0; i < nsem; ++i) {
      f << std::scientific << xsem[i] << "\t" << ysem[i] << "\t" << zsem[i]
        << "\t" << eps_sem_x[i] << "\t" << eps_sem_y[i] << "\t" << eps_sem_z[i]
        << "\n";
    }

    f.close();
  }
}

template <typename T> void SEM<T>::read_sem() {
  std::string fname = "sem.dat";
  std::fstream f;
  f.open(fname, std::ios::in);
  if (f.is_open()) {
    f >> nsem;
    for (auto i = 0; i < nsem; ++i)
      f >> std::scientific >> xsem[i] >> ysem[i] >> zsem[i] >> eps_sem_x[i] >>
          eps_sem_y[i] >> eps_sem_z[i];
    f.close();
  }
}

template <typename T> void SEM<T>::update_sem(const T dt) {
  std::vector<T> rnd;
  rnd.assign(5, 0.0);
  size_t threads_per_block = 1024;
  size_t number_of_blocks = (nsem + threads_per_block -1) /threads_per_block;

  hipError_t err;
  hiprandState *devStates;
  hipMalloc((void **)&devStates,
             threads_per_block * number_of_blocks * sizeof(hiprandState));

  setup_kernel<<<number_of_blocks, threads_per_block>>>(devStates);

  update_sem_gpu<<<number_of_blocks, threads_per_block>>>(
      _gpu.xsem, _gpu.ysem, _gpu.zsem, _gpu.eps_sem_x, _gpu.eps_sem_y,
      _gpu.eps_sem_z, STGInputs::ub0, dt, sigma_max, ly_sem, lz_sem, devStates,
      nsem);
  err = hipGetLastError();
  if (err != hipSuccess)
    printf("Error : %s \n", hipGetErrorString(err));

  hipDeviceSynchronize();
  hipFree(devStates);
}

template <typename T>
void SEM<T>::get_sem_fluc(const std::vector<T> &xx, const T tke, const T eps,
                          std::vector<T> &uvecSEM) {
  std::vector<T> Rij, aij;

  Rij.assign(6, 0.0);
  aij.assign(9, 0.0);


  STGUtils::get_re_stress(tke, Rij);

  STGUtils::re_cholesky_decomp(Rij, aij);

  T sigma = STGUtils::get_sigma(tke, eps, STGInputs::delta0, xx[3]);
  T uSEM = 0.0;
  T vSEM = 0.0;
  T wSEM = 0.0;
  hipError_t err;

  hipMemcpy(_gpu.a_ij, aij.data(), 9 * sizeof(T), hipMemcpyHostToDevice);
  hipMemcpy(_gpu.xx, xx.data(), 3 * sizeof(T), hipMemcpyHostToDevice);

  hipMemset(_gpu.uvec, 0, 3 * sizeof(T));

  size_t threads_per_block = 256;  //1024;
  size_t number_of_blocks = 32; //(nsem +threads_per_block -1)/threads_per_block;

  get_sem_fluc_gpu<<<number_of_blocks, threads_per_block>>>(
      _gpu.xsem, _gpu.ysem, _gpu.zsem, _gpu.eps_sem_x, _gpu.eps_sem_y,
      _gpu.eps_sem_z, _gpu.uvec, sigma, _gpu.xx, _gpu.a_ij, nsem);

  err = hipGetLastError();
  if (err != hipSuccess)
    printf("Error : %s \n", hipGetErrorString(err));

  hipDeviceSynchronize();

  hipMemcpy(uvecSEM.data(), _gpu.uvec, 3 * sizeof(T), hipMemcpyDeviceToHost);

  uvecSEM[0] = uvecSEM[0] *
               std::sqrt((lx_sem * ly_sem * lz_sem) / std::pow(sigma, 3)) /
               std::sqrt(nsem * 1.0);
  uvecSEM[1] = uvecSEM[1] *
               std::sqrt((lx_sem * ly_sem * lz_sem) / std::pow(sigma, 3)) /
               std::sqrt(nsem * 1.0);
  uvecSEM[2] = uvecSEM[2] *
               std::sqrt((lx_sem * ly_sem * lz_sem) / std::pow(sigma, 3)) /
               std::sqrt(nsem * 1.0);
}

template <typename T> void SEM<T>::info() {
  std::cout << "\n"
            << "SEM information"
            << "\n";
  std::cout << "-----------------------------";
  std::cout << "No. of SEM pts.:"
            << "\t" << nsem << "\n";
}

template <typename T> SEM<T>::~SEM() {
  size_t size = nsem * sizeof(T);

  hipMemcpy(xsem.data(), _gpu.xsem, size, hipMemcpyDeviceToHost);
  hipMemcpy(ysem.data(), _gpu.ysem, size, hipMemcpyDeviceToHost);
  hipMemcpy(zsem.data(), _gpu.zsem, size, hipMemcpyDeviceToHost);
  hipMemcpy(eps_sem_x.data(), _gpu.eps_sem_x, size, hipMemcpyDeviceToHost);
  hipMemcpy(eps_sem_y.data(), _gpu.eps_sem_y, size, hipMemcpyDeviceToHost);
  hipMemcpy(eps_sem_z.data(), _gpu.eps_sem_z, size, hipMemcpyDeviceToHost);


  write_sem();
  // Deallocate  GPU field variables
  std::cout << "Deallocation GPU" << std::endl;
  hipFree(_gpu.xsem);
  hipFree(_gpu.ysem);
  hipFree(_gpu.zsem);
  hipFree(_gpu.uvec);
  hipFree(_gpu.eps_sem_x);
  hipFree(_gpu.eps_sem_y);
  hipFree(_gpu.eps_sem_z);
  hipFree(_gpu.a_ij);
  hipFree(_gpu.xx);
}
